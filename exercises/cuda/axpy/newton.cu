#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.h"

__host__
double f(double x) {
    return exp(cos(x))-2;
};

__host__
double fp(double x) {
    return -sin(x) * exp(cos(x));
};

// implements newton solve for
//      f(x) = 0
// where
//      f(x) = exp(cos(x)) - 2
void newton_host(int n, double *x) {
    for(int i=0; i<n; ++i) {
        auto x0 = x[i];
        for(int iter=0; iter<5; ++iter) {
            x0 -= f(x0)/fp(x0);
        }
        x[i] = x0;
    }
}

// TODO : implement newton_device() kernel that performs the work in newton_host
//        in parallel on the GPU

int main(int argc, char** argv) {
    size_t pow        = read_arg(argc, argv, 1, 20);

    size_t N = 1 << pow;
    auto size_in_bytes = N * sizeof(double);

    std::cout << "memory copy overlap test of length N = " << N
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << std::endl;

    hipInit(0);

    double* xd = malloc_device<double>(N);
    double* xh = malloc_host<double>(N, 1.5);
    double* x  = malloc_host<double>(N);

    // compute kernel launch configuration
    auto block_dim = 128;
    auto grid_dim = (N+block_dim-1)/block_dim;

    auto time_h2d = -get_time();
    copy_to_device(xh, xd, N);
    time_h2d += get_time();

    hipDeviceSynchronize();
    auto time_kernel = -get_time();
    // TODO: launch kernel (use block_dim and grid_dim calculated above)
    hipDeviceSynchronize();
    time_kernel += get_time();

    auto time_d2h = -get_time();
    copy_to_host(xd, x, N);
    time_d2h += get_time();

    std::cout << "-------\ntimings\n-------" << std::endl;
    std::cout << "H2D    : " << time_h2d    << std::endl;
    std::cout << "D2H    : " << time_d2h    << std::endl;
    std::cout << "kernel : " << time_kernel << std::endl;

    // check for errors
    auto errors = 0;
    for(auto i=0; i<N; ++i) {
        if(std::fabs(f(x[i]))>1e-10) {
            errors++;
        }
    }
    if(errors>0) std::cout << "\n============ FAILED with " << errors << " errors" << std::endl;
    else         std::cout << "\n============ PASSED" << std::endl;

    hipFree(xd);
    free(xh);
    free(x);

    return 0;
}

