#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.h"

int main(int argc, char** argv) {
    size_t pow = read_arg(argc, argv, 1, 16);
    size_t n = 1 << pow;
    auto size_in_bytes = n * sizeof(double);

    std::cout << "memcopy and daxpy test of length n = " << n
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << std::endl;

    hipInit(0);

    // initialize cublas
    auto cublas_handle = get_cublas_handle();

    double* x_device = malloc_device<double>(n);
    double* y_device = malloc_device<double>(n);

    double* x_host = malloc_host<double>(n, 1.5);
    double* y_host = malloc_host<double>(n, 3.0);
    double* y      = malloc_host<double>(n, 0.0);

    copy_to_device<double>(x_host, x_device, n);

    // copy to device
    auto start = get_time();
    copy_to_device<double>(x_host, x_device, n);
    copy_to_device<double>(y_host, y_device, n);
    auto time_H2D = get_time() - start;

    // y = y + 2 * x
    start = get_time();
    double alpha = 2.0;
    auto cublas_status =
        hipblasDaxpy(cublas_handle, n, &alpha, x_device, 1, y_device, 1);
    hipDeviceSynchronize();
    auto time_axpy = get_time() - start;

    // copy result back to host
    start = get_time();
    copy_to_host<double>(y_device, y, n);
    auto time_D2H = get_time() - start;

    std::cout << "-------\ntimings\n-------" << std::endl;
    std::cout << "H2D  : " << time_H2D << std::endl;
    std::cout << "D2H  : " << time_D2H << std::endl;
    std::cout << "axpy : " << time_axpy << std::endl;

    // check for errors
    auto errors = 0;
    #pragma omp parallel for reduction(+:errors)
    for(auto i=0; i<n; ++i) {
        if(std::fabs(6.-y[i])>1e-15) {
            errors++;
        }
    }

    if(errors>0) {
        std::cout << "\n============ FAILED with "
                  << errors << " errors" << std::endl;
    }
    else {
        std::cout << "\n============ PASSED" << std::endl;
    }

    hipFree(x_device);
    hipFree(y_device);

    free(x_host);
    free(y_host);
    free(y);

    return 0;
}

