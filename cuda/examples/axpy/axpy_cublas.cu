#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>

#include "util.h"

int main(int argc, char** argv) {
    size_t N = read_arg(argc, argv, 1, 16);
    auto size_in_bytes = N * sizeof(double);

    std::cout << "memcopy and daxpy test of length N = " << N
              << " : " << size_in_bytes/(1024.*1024.) << "MB"
              << std::endl;

    hipInit(0);

    // initialize cublas
    auto cublas_handle = get_cublas_handle();

    double* x_device = malloc_device<double>(N);
    double* y_device = malloc_device<double>(N);

    double* x_host = malloc_host<double>(N, 1.5);
    double* y_host = malloc_host<double>(N, 3.0);
    double* y      = malloc_host<double>(N, 0.0);

    // copy to device
    auto start = get_time();
    copy_to_device<double>(x_host, x_device, N);
    copy_to_device<double>(y_host, y_device, N);
    auto time_H2D = get_time() - start;

    // y += 2 * x
    start = get_time();
    double alpha = 2.0;
    auto cublas_status =
        hipblasDaxpy(cublas_handle, N, &alpha, x_device, 1, y_device, 1);
    hipDeviceSynchronize();
    auto time_axpy = get_time() - start;

    // copy result back to host
    start = get_time();
    copy_to_host<double>(y_device, y, N);
    auto time_D2H = get_time() - start;

    std::cout << "-------\ntimings\n-------" << std::endl;
    std::cout << "H2D  : " << time_H2D << std::endl;
    std::cout << "D2H  : " << time_D2H << std::endl;
    std::cout << "axpy : " << time_axpy << std::endl;

    // check for errors
    auto errors = 0;
    #pragma omp parallel for reduction(+:errors)
    for(auto i=0; i<N; ++i) {
        if(std::fabs(6.-y[i])>1e-15) {
            errors++;
        }
    }

    if(errors>0) {
        std::cout << "\n============ FAILED with "
                  << errors << " errors" << std::endl;
    }
    else {
        std::cout << "\n============ PASSED" << std::endl;
    }

    hipFree(x_device);
    hipFree(y_device);

    free(x_host);
    free(y_host);
    free(y);

    return 0;
}

